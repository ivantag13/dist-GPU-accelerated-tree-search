#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"
{
#endif

#include <stdlib.h>
#include "PFSP_gpu_lib.cuh"
#include "bounds_gpu.cu"

// CUDA error checking
// TODO: fix portability for variable hipError_t (https://rocm.docs.amd.com/projects/HIP/en/docs-develop/how-to/hip_porting_guide.html)
// #define gpuErrchk(ans)                          \
//   {                                             \
//     gpuAssert((ans), __FILE__, __LINE__, true); \
//   }
//   void gpuAssert(hipError_t code, const char *file, int line, bool abort)
//   {
//     if (code != hipSuccess)
//     {
//       fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort)
//         exit(code);
//     }
//   }

  __device__ void swap_cuda(int *a, int *b)
  {
    int tmp = *b;
    *b = *a;
    *a = tmp;
  }

  void printDims(dim3 gridDim, dim3 blockDim)
  {
    printf("Grid Dimensions : [%d, %d, %d] blocks. \n",
           gridDim.x, gridDim.y, gridDim.z);

    printf("Block Dimensions : [%d, %d, %d] threads.\n",
           blockDim.x, blockDim.y, blockDim.z);
  }

  // Evaluate a bulk of parent nodes on GPU using lb1
  __global__ void evaluate_gpu_lb1(const int jobs, const int size, Node *parents_d, const int parentsSize,
                                   const lb1_bound_data lbound1_d, int *bounds, int *sumOffSets_d, int *nodeIndex)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < size)
    {
      int parentId = nodeIndex[threadId];
      Node parent = parents_d[parentId];
      int depth = parent.depth;
      int limit1 = parent.limit1;
      int k = threadId + depth;
      if (parentId != 0)
        k -= sumOffSets_d[parentId - 1];

      swap_cuda(&parent.prmu[depth], &parent.prmu[k]);
      lb1_bound_gpu(lbound1_d, parent.prmu, limit1 + 1, jobs, &bounds[threadId]);
      // swap_cuda(&parent.prmu[depth], &parent.prmu[k]);
    }
  }

  /*
    NOTE: This lower bound evaluates all the children of a given parent at the same time.
    Therefore, the GPU loop is on the parent nodes and not on the children ones, in contrast
    to the other lower bounds.
  */
  // Evaluate a bulk of parent nodes on GPU using lb1_d.
  __global__ void evaluate_gpu_lb1_d(const int jobs, const int size, Node *parents_d, const lb1_bound_data lbound1_d, int *bounds)
  {
    int parentId = blockIdx.x * blockDim.x + threadIdx.x;

    if (parentId < size)
    {
      Node parent = parents_d[parentId];

      // Vector of integers of size MAX_JOBS
      int lb_begin[MAX_JOBS];

      lb1_children_bounds_gpu(lbound1_d, parent.prmu, parent.limit1, jobs, lb_begin);

      for (int k = 0; k < jobs; k++)
      {
        if (k >= parent.limit1 + 1)
        {
          const int job = parent.prmu[k];
          bounds[parentId * jobs + k] = lb_begin[job];
        }
      }
    }
  }

  // Evaluate a bulk of parent nodes on GPU using lb2.
  __global__ void evaluate_gpu_lb2(const int jobs, const int size, int best, Node *parents_d, int parentsSize, const lb1_bound_data lbound1_d,
                                   const lb2_bound_data lbound2_d, int *bounds, int *sumOffSets_d, int *nodeIndex_d)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < size)
    {
      int parentId = nodeIndex_d[threadId];
      Node parent = parents_d[parentId];
      int depth = parent.depth;
      int limit1 = parent.limit1;
      int k = threadId + depth;
      if (parentId != 0)
        k -= sumOffSets_d[parentId - 1];

      swap_cuda(&parent.prmu[depth], &parent.prmu[k]);
      lb2_bound_gpu(lbound1_d, lbound2_d, parent.prmu, limit1 + 1, jobs, best, &bounds[threadId]);
      // swap_cuda(&parent.prmu[depth], &parent.prmu[k]);
    }
  }

  void evaluate_gpu(const int jobs, const int lb, const int size, const int nbBlocks, const int parentsSize, int *best,
                    const lb1_bound_data lbound1, const lb2_bound_data lbound2, Node *parents, int *bounds, int *sumOffSets_d, int *nodeIndex_d)
  {
    // 1D grid of 1D nbBlocks(_lb1_d) blocks with block size BLOCK_SIZE
    int nbBlocks_lb1_d;
    switch (lb)
    {
    case 0: // lb1_d
      nbBlocks_lb1_d = ceil((double)nbBlocks / jobs);
      evaluate_gpu_lb1_d<<<nbBlocks_lb1_d, BLOCK_SIZE>>>(jobs, size, parents, lbound1, bounds);
      return;
      break;

    case 1: // lb1
      evaluate_gpu_lb1<<<nbBlocks, BLOCK_SIZE>>>(jobs, size, parents, parentsSize, lbound1, bounds, sumOffSets_d, nodeIndex_d);
      return;
      break;

    case 2: // lb2
      evaluate_gpu_lb2<<<nbBlocks, BLOCK_SIZE>>>(jobs, size, *best, parents, parentsSize, lbound1, lbound2, bounds, sumOffSets_d, nodeIndex_d);
      return;
      break;
    }
  }

  void lb1_alloc_gpu(lb1_bound_data *lbound1_d, lb1_bound_data *lbound1, int *p_times_d, int *min_heads_d, int *min_tails_d, int jobs, int machines)
  {
    // Allocating and copying memory necessary for deep copy of lbound1
    hipMalloc((void **)&p_times_d, jobs * machines * sizeof(int));
    hipMalloc((void **)&min_heads_d, machines * sizeof(int));
    hipMalloc((void **)&min_tails_d, machines * sizeof(int));
    hipMemcpy(p_times_d, lbound1->p_times, (jobs * machines) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(min_heads_d, lbound1->min_heads, machines * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(min_tails_d, lbound1->min_tails, machines * sizeof(int), hipMemcpyHostToDevice);

    // Deep copy of lbound1
    lbound1_d->p_times = p_times_d;
    lbound1_d->min_heads = min_heads_d;
    lbound1_d->min_tails = min_tails_d;
    lbound1_d->nb_jobs = lbound1->nb_jobs;
    lbound1_d->nb_machines = lbound1->nb_machines;

    return;
  }

  void lb2_alloc_gpu(lb2_bound_data *lbound2_d, lb2_bound_data *lbound2, int *johnson_schedule_d, int *lags_d,
                     int *machine_pairs_1_d, int *machine_pairs_2_d, int *machine_pair_order_d, int jobs, int machines)
  {
    // Allocating and copying memory necessary for deep copy of lbound2
    int nb_mac_pairs = lbound2->nb_machine_pairs;
    hipMalloc((void **)&johnson_schedule_d, (nb_mac_pairs * jobs) * sizeof(int));
    hipMalloc((void **)&lags_d, (nb_mac_pairs * jobs) * sizeof(int));
    hipMalloc((void **)&machine_pairs_1_d, nb_mac_pairs * sizeof(int));
    hipMalloc((void **)&machine_pairs_2_d, nb_mac_pairs * sizeof(int));
    hipMalloc((void **)&machine_pair_order_d, nb_mac_pairs * sizeof(int));
    hipMemcpy(johnson_schedule_d, lbound2->johnson_schedules, (nb_mac_pairs * jobs) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(lags_d, lbound2->lags, (nb_mac_pairs * jobs) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(machine_pairs_1_d, lbound2->machine_pairs_1, nb_mac_pairs * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(machine_pairs_2_d, lbound2->machine_pairs_2, nb_mac_pairs * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(machine_pair_order_d, lbound2->machine_pair_order, nb_mac_pairs * sizeof(int), hipMemcpyHostToDevice);

    // Deep copy of lbound2
    lbound2_d->johnson_schedules = johnson_schedule_d;
    lbound2_d->lags = lags_d;
    lbound2_d->machine_pairs_1 = machine_pairs_1_d;
    lbound2_d->machine_pairs_2 = machine_pairs_2_d;
    lbound2_d->machine_pair_order = machine_pair_order_d;
    lbound2_d->nb_machine_pairs = lbound2->nb_machine_pairs;
    lbound2_d->nb_jobs = lbound2->nb_jobs;
    lbound2_d->nb_machines = lbound2->nb_machines;
    return;
  }

#ifdef __cplusplus
}
#endif
